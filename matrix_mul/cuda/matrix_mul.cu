#include "hip/hip_runtime.h"
/*
    Copyright (C) 2011  Abhinav Jauhri (abhinav.jauhri@gmail.com), Carnegie Mellon University - Silicon Valley 

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matrix_mul.h"
#define TILE_SZ 4 
#define BLK_SZ 16

namespace cuda
{
  __global__ 
  void 
  matrix_mul_kernel(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, int sq_dimension)
  {
    int i,j,k;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int col = blockIdx.x * TILE_SZ * BLK_SZ + threadIdx.x;
    int row = blockIdx.y * BLK_SZ + threadIdx.y;

    __shared__ float mat1[BLK_SZ][BLK_SZ];
    __shared__ float mat2[BLK_SZ][BLK_SZ*TILE_SZ]; 

    float sum[TILE_SZ];

    for (j = 0; j < TILE_SZ; j ++)
	sum[j] = 0.0;
   
    for (k = 0; k < (sq_dimension + BLK_SZ - 1)/BLK_SZ; k ++)
    {
    	if (k*BLK_SZ + tx < sq_dimension && row < sq_dimension)
    	{
    		mat1[ty][tx] = sq_matrix_1[row*sq_dimension + k * BLK_SZ + tx];
    	}
    	else
    	{
    		mat1[ty][tx] = 0.0;
    	}

	for (j = 0; j < TILE_SZ; j ++)
	{
		if (k*BLK_SZ + ty < sq_dimension && col + j*BLK_SZ < sq_dimension)
		{
    			mat2[ty][tx + j*BLK_SZ] = sq_matrix_2[(ty + k*BLK_SZ)*sq_dimension + col + j*BLK_SZ];
		}
		else
		{
    			mat2[ty][tx + j*BLK_SZ] = 0.0;
		}
	}

		
	__syncthreads();
	for (j = 0; j < TILE_SZ; j ++)
	{
    		for(i = 0; i < BLK_SZ; i++)
    		{
			sum[j] += mat1[ty][i]*mat2[i][tx + j*BLK_SZ];
    		}
	}
	__syncthreads();
    }

    for (j = 0; j < TILE_SZ; j ++)
    {
    	if (row < sq_dimension && col + j*BLK_SZ <  sq_dimension)	
    		sq_matrix_result[row*sq_dimension + col + j*BLK_SZ] = sum[j];
    }
   
  }
  
  void 
  matrix_multiplication(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, unsigned int sq_dimension)
  {
    int size = sq_dimension * sq_dimension * sizeof(float);
    float *sq_matrix_1_d, *sq_matrix_2_d, *sq_matrix_result_d;
    
    /***************************************************
  1st Part: Allocation of memory on device memory  
    ****************************************************/
    
    /* copy sq_matrix_1 and sq_matrix_2 to device memory */
    hipMalloc((void**) &sq_matrix_1_d, size);
    hipMemcpy(sq_matrix_1_d, sq_matrix_1, size, hipMemcpyHostToDevice);
    hipMalloc((void**) &sq_matrix_2_d, size);
    hipMemcpy(sq_matrix_2_d, sq_matrix_2, size, hipMemcpyHostToDevice);
    
    /*allocate sq_matrix_result on host */
    hipMalloc((void**) &sq_matrix_result_d, size);
    
    /***************************************************
   2nd Part: Inovke kernel 
    ****************************************************/
    dim3 dimBlock(BLK_SZ, BLK_SZ);
    const unsigned int dimY = (sq_dimension + BLK_SZ - 1)/BLK_SZ;
    const unsigned int dimX = (sq_dimension + TILE_SZ*BLK_SZ - 1)/(TILE_SZ*BLK_SZ);
    dim3 dimGrid(dimX, dimY);
    matrix_mul_kernel<<<dimGrid, dimBlock, BLK_SZ*BLK_SZ*sizeof(float) + TILE_SZ*BLK_SZ*BLK_SZ*sizeof(float)>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
   
    /***************************************************
   3rd Part: Transfer result from device to host 
    ****************************************************/
    hipMemcpy(sq_matrix_result, sq_matrix_result_d, size, hipMemcpyDeviceToHost);
    hipFree(sq_matrix_1_d);
    hipFree(sq_matrix_2_d);
    hipFree(sq_matrix_result_d);
  }  
} // namespace cuda
